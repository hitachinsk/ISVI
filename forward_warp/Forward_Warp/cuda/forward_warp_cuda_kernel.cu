#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <THC/THCAtomics.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

#include "forward_warp.h"
using at::native::detail::GridSamplerInterpolation;

static __forceinline__ __device__ 
int get_im_index(
    const int b,
    const int c,
    const int h,
    const int w,
    const size_t C,
    const size_t H,
    const size_t W) {
  return b*C*H*W + c*H*W + h*W + w;
}

template <typename scalar_t>
__global__ void forward_warp_cuda_forward_kernel(
    const int total_step,
    const scalar_t* im0,
    const scalar_t* flow,
    scalar_t* im1,
    const int B,
    const int C,
    const int H,
    const int W,
    const GridSamplerInterpolation interpolation_mode) {
  // CUDA_KERNEL_LOOP(index, total_step-1) {
  // bug fix, thx to @tkkcc
  CUDA_KERNEL_LOOP(index, total_step) {
    const int b = index / (H * W);
    const int h = (index-b*H*W) / W;
    const int w = index % W;
    const scalar_t x = (scalar_t)w + flow[index*2+0];
    const scalar_t y = (scalar_t)h + flow[index*2+1];
    if (interpolation_mode == GridSamplerInterpolation::Bilinear) {
      const int x_f = static_cast<int>(::floor(x));
      const int y_f = static_cast<int>(::floor(y));
      const int x_c = x_f + 1;
      const int y_c = y_f + 1;
      if(x_f>=0 && x_c<W && y_f>=0 && y_c<H){
        const scalar_t nw_k = (x_c - x) * (y_c - y);
        const scalar_t ne_k = (x - x_f) * (y_c - y);
        const scalar_t sw_k = (x_c - x) * (y - y_f);
        const scalar_t se_k = (x - x_f) * (y - y_f);
        const scalar_t* im0_p = im0+get_im_index(b, 0, h, w, C, H, W);
        scalar_t* im1_p = im1+get_im_index(b, 0, y_f, x_f, C, H, W);
        for (int c = 0; c < C; ++c, im0_p+=H*W, im1_p+=H*W){
            atomicAdd(im1_p,     nw_k*(*im0_p));
            atomicAdd(im1_p+1,   ne_k*(*im0_p));
            atomicAdd(im1_p+W,   sw_k*(*im0_p));
            atomicAdd(im1_p+W+1, se_k*(*im0_p));
        }
      }
    } 
    else if (interpolation_mode == GridSamplerInterpolation::Nearest) {
      const int x_nearest = static_cast<int>(::round(x));
      const int y_nearest = static_cast<int>(::round(y));
      if(x_nearest>=0 && x_nearest<W && y_nearest>=0 && y_nearest<H){
        const scalar_t* im0_p = im0+get_im_index(b, 0, h, w, C, H, W);
        scalar_t* im1_p = im1+get_im_index(b, 0, y_nearest, x_nearest, C, H, W);
        for (int c = 0; c < C; ++c, im0_p += H*W, im1_p += H*W) {
            *im1_p = *im0_p;
        }
      }
    }
  }
}

template <typename scalar_t>
__global__ void forward_warp_cuda_backward_kernel(
    const int total_step,
    const scalar_t* grad_output,
    const scalar_t* im0,
    const scalar_t* flow,
    scalar_t* im0_grad,
    scalar_t* flow_grad,
    const int B,
    const int C,
    const int H,
    const int W,
    const GridSamplerInterpolation interpolation_mode) {
  CUDA_KERNEL_LOOP(index, total_step) {
    const int b = index / (H * W);
    const int h = (index-b*H*W) / W;
    const int w = index % W;
    const scalar_t x = (scalar_t)w + flow[index*2+0];
    const scalar_t y = (scalar_t)h + flow[index*2+1];
    if (interpolation_mode == GridSamplerInterpolation::Bilinear) {
      const int x_f = static_cast<int>(::floor(x));
      const int y_f = static_cast<int>(::floor(y));
      const int x_c = x_f + 1;
      const int y_c = y_f + 1;
      if(x_f>=0 && x_c<W && y_f>=0 && y_c<H){
        const scalar_t nw_k = (x_c - x) * (y_c - y);
        const scalar_t sw_k = (x_c - x) * (y - y_f);
        const scalar_t ne_k = (x - x_f) * (y_c - y);
        const scalar_t se_k = (x - x_f) * (y - y_f);
        scalar_t flow_grad_x = 0;
        scalar_t flow_grad_y = 0;
        scalar_t* im0_grad_p = im0_grad+get_im_index(b, 0, h, w, C, H, W);
        for (int c = 0; c < C; ++c, im0_grad_p+=H*W){
          const scalar_t nw_grad = grad_output[get_im_index(b, c, y_f, x_f, C, H, W)];
          const scalar_t ne_grad = grad_output[get_im_index(b, c, y_f, x_c, C, H, W)];
          const scalar_t sw_grad = grad_output[get_im_index(b, c, y_c, x_f, C, H, W)];
          const scalar_t se_grad = grad_output[get_im_index(b, c, y_c, x_c, C, H, W)];
          const scalar_t p = im0[get_im_index(b, c, h, w, C, H, W)];
          atomicAdd(im0_grad_p, nw_k*nw_grad);
          atomicAdd(im0_grad_p, ne_k*ne_grad);
          atomicAdd(im0_grad_p, sw_k*sw_grad);
          atomicAdd(im0_grad_p, se_k*se_grad);
          flow_grad_x -= (y_c-y)*p*nw_grad;
          flow_grad_y -= (x_c-x)*p*nw_grad;
          flow_grad_x += (y_c-y)*p*ne_grad;
          flow_grad_y -= (x-x_f)*p*ne_grad;
          flow_grad_x -= (y-y_f)*p*sw_grad;
          flow_grad_y += (x_c-x)*p*sw_grad;
          flow_grad_x += (y-y_f)*p*se_grad;
          flow_grad_y += (x-x_f)*p*se_grad;
        }
        flow_grad[index*2+0] = flow_grad_x;
        flow_grad[index*2+1] = flow_grad_y;
      }
    } 
    else if (interpolation_mode == GridSamplerInterpolation::Nearest) {
      const int x_nearest = static_cast<int>(::round(x));
      const int y_nearest = static_cast<int>(::round(y));
      if(x_nearest>=0 && x_nearest<W && y_nearest>=0 && y_nearest<H){
        scalar_t* im0_grad_p = im0_grad+get_im_index(b, 0, h, w, C, H, W);
        const scalar_t* im1_grad_p = grad_output+get_im_index(b, 0, y_nearest, x_nearest, C, H, W);
        for (int c = 0; c < C; ++c, im0_grad_p += H*W, im1_grad_p += H*W) {
            *im0_grad_p = *im1_grad_p;
        }
      }
    }
  }
}

at::Tensor forward_warp_cuda_forward(
    const at::Tensor im0, 
    const at::Tensor flow,
    const GridSamplerInterpolation interpolation_mode) {
  auto im1 = at::zeros_like(im0);
  const int B = im0.size(0);
  const int C = im0.size(1);
  const int H = im0.size(2);
  const int W = im0.size(3);
  const int total_step = B * H * W;
  AT_DISPATCH_FLOATING_TYPES(im0.scalar_type(), "forward_warp_forward_cuda", ([&] {
    forward_warp_cuda_forward_kernel<scalar_t>
    <<<GET_BLOCKS(total_step), CUDA_NUM_THREADS>>>(
      total_step,
      im0.data<scalar_t>(),
      flow.data<scalar_t>(),
      im1.data<scalar_t>(),
      B, C, H, W,
      interpolation_mode);
  }));

  return im1;
}

std::vector<at::Tensor> forward_warp_cuda_backward(
    const at::Tensor grad_output,
    const at::Tensor im0, 
    const at::Tensor flow,
    const GridSamplerInterpolation interpolation_mode) {
  auto im0_grad = at::zeros_like(grad_output);
  auto flow_grad = at::empty_like(flow);
  const int B = im0.size(0);
  const int C = im0.size(1);
  const int H = im0.size(2);
  const int W = im0.size(3);
  const int total_step = B * H * W;

  AT_DISPATCH_FLOATING_TYPES(grad_output.type(), "forward_warp_backward_cuda", ([&] {
    forward_warp_cuda_backward_kernel<scalar_t>
    <<<GET_BLOCKS(total_step), CUDA_NUM_THREADS>>>(
      total_step,
      grad_output.data<scalar_t>(),
      im0.data_ptr<scalar_t>(),
      flow.data<scalar_t>(),
      im0_grad.data<scalar_t>(),
      flow_grad.data<scalar_t>(),
      B, C, H, W,
      interpolation_mode);
  }));

  return {im0_grad, flow_grad};
}
